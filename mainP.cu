#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "support.h"

__global__ void kernelP(int m, int n, int k, 
                        const float *A, const float *B, float* C) 
{
	const unsigned int BLOCK_SIZE = 32;
	
	int bx =  blockIdx.x; int by =  blockIdx.y;  
	int tx = threadIdx.x; int ty = threadIdx.y; 
	
	int Row = by * BLOCK_SIZE + ty;
	int Col = bx * BLOCK_SIZE + tx;
	 
	if (Row < m && Col < n)
		for(unsigned int i = 0; i < k; ++i)
			C[row*n + col] += A[row*k + i]*B[i*n + col];
	__syncthreads();
}

void midP(char transa, char transb, \
		  int m, int n, int k, \
		  float alpha, \
		  const float *A, int lda, \
		  const float *B, int ldb, \
		  float beta, \
		 float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
		printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
		printf("unsupported value of 'transb'\n");
		return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
		printf("unsupported value of alpha\n");
		return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
		printf("unsupported value of beta\n");
		return;
    }

    const unsigned int BLOCK_SIZE = 32;

	unsigned int grid_y = (unsigned int) ceil((double)m / (double)BLOCK_SIZE); 
	unsigned int grid_x = (unsigned int) ceil((double)n / (double)BLOCK_SIZE); 
	dim3 gridDim(grid_x, grid_y); 
	dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
	
	kernelP<<<gridDim, blockDim>>>(m, n, k, A, B, C);
}

int main (int argc, char *argv[])
{

    Timer timer;
    printf("\nRunning Non-Tiled..."); fflush(stdout);
    startTime(&timer);

    float *A_h, *B_h, *C_h;
    float *A_d, *B_d, *C_d;
    size_t A_sz, B_sz, C_sz;
    unsigned matArow, matAcol;
    unsigned matBrow, matBcol;
    dim3 dim_grid, dim_block;

    if (argc == 1) {
        matArow = 1000;
        matAcol = matBrow = 1000;
        matBcol = 1000;
    } else if (argc == 2) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[1]);
        matBcol = atoi(argv[1]);
    } else if (argc == 4) {
        matArow = atoi(argv[1]);
        matAcol = matBrow = atoi(argv[2]);
        matBcol = atoi(argv[3]);
    } else {
        printf("\n    Invalid input parameters!"
           "\n    Usage: ./sgemm                # All matrices are 1000 x 1000"
           "\n    Usage: ./sgemm <m>            # All matrices are m x m"
           "\n    Usage: ./sgemm <m> <k> <n>    # A: m x k, B: k x n, C: m x n"
           "\n");
        exit(0);
    }

    A_sz = matArow*matAcol;
    B_sz = matBrow*matBcol;
    C_sz = matArow*matBcol;

    A_h = (float*) malloc( sizeof(float)*A_sz );
    for (unsigned int i=0; i < A_sz; i++) A_h[i] = (rand()%100)/100.00;

    B_h = (float*) malloc( sizeof(float)*B_sz );
    for (unsigned int i=0; i < B_sz; i++) B_h[i] = (rand()%100)/100.00;

    C_h = (float*) malloc( sizeof(float)*C_sz );

	printf("    A: %u x %u\n    B: %u x %u\n    C: %u x %u\n", 
		   matArow, matAcol, matBrow, matBcol, matArow, matBcol);
	
	hipMalloc((void **) &A_d, sizeof(float)*A_sz);
	hipMalloc((void **) &B_d, sizeof(float)*B_sz);
	hipMalloc((void **) &C_d, sizeof(float)*C_sz);
	hipDeviceSynchronize();

	hipMemcpy(A_d, A_h, sizeof(float)*A_sz, hipMemcpyHostToDevice);
	hipMemcpy(B_d, B_h, sizeof(float)*B_sz, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	midP('N', 'N', matArow, matBcol, matBrow, 1.0f,
		 A_d, matArow, B_d, matBrow, 0.0f, C_d, matBrow);
    hipDeviceSynchronize();

    hipMemcpy(C_h, C_d, sizeof(float)*C_sz, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
	
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));

    printf("Verifying results..."); fflush(stdout);

    verify(A_h, B_h, C_h, matArow, matAcol, matBcol);

    free(A_h);
    free(B_h);
    free(C_h);

	hipFree(A_d);
	hipFree(B_d);
	hipFree(C_d);

	return 0;
}